#include "hip/hip_runtime.h"
#include <iostream>
#include <torch/torch.h>

#include "sparse_matrix_utils.hpp"
#include "cuda_utils.hpp"

#define ROWS            16384
#define COLUMNS         16384
#define SPARSITY_RATIO  0.2
#define BLOCK_SIZE      1024
#define PAD_VAL         (1<<20)

template <typename T>
void move_ell_matrix_to_device(ELLMatrix<T>& h_matrix, ELLMatrix<T>& d_matrix) {
    // Allocate memory on the device
    CHECK_CUDA_ERROR(hipMalloc(&d_matrix.rowPtrs, (h_matrix.R+1) * sizeof(unsigned int)));
    CHECK_CUDA_ERROR(hipMalloc(&d_matrix.colIdx, (h_matrix.R * h_matrix.max_nz_in_row) * sizeof(unsigned int)));
    CHECK_CUDA_ERROR(hipMalloc(&d_matrix.value, (h_matrix.R * h_matrix.max_nz_in_row) * sizeof(T)));

    // Set dimensions
    d_matrix.max_nz_in_row = h_matrix.max_nz_in_row;
    d_matrix.R = h_matrix.R;
    d_matrix.C = h_matrix.C;
    d_matrix.num_nonzero = h_matrix.num_nonzero;

    // Copy data from host to device
    CHECK_CUDA_ERROR(hipMemcpy(d_matrix.rowPtrs, h_matrix.rowPtrs, (h_matrix.R+1) * sizeof(unsigned int), hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(d_matrix.colIdx, h_matrix.colIdx, (h_matrix.R * h_matrix.max_nz_in_row) * sizeof(unsigned int), hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(d_matrix.value, h_matrix.value, (h_matrix.R * h_matrix.max_nz_in_row) * sizeof(T), hipMemcpyHostToDevice));
}

template <typename T>
__global__ void spmv_ell_kernel(ELLMatrix<T> A, T* x, T* y) {
    unsigned int row = blockIdx.x * blockDim.x + threadIdx.x;

    if(row < A.R) {
        float row_sum = 0.0f;
        for(unsigned int i=0; i<A.max_nz_in_row; i++) {
            unsigned int idx = i * A.R + row;
            unsigned int col_idx = A.colIdx[idx];
            if(col_idx != static_cast<T>(PAD_VAL)) {
                T value = A.value[idx];
                row_sum += value * x[col_idx];
            }
        }
        y[row] += row_sum;
    }
}

inline unsigned int cdiv(unsigned int a, unsigned int b) {
    return (a + b - 1)/b;
}

template <typename T>
void spmv_ell(ELLMatrix<T> A, T* x, T* y) {
    dim3 blockSize(BLOCK_SIZE);
    dim3 gridSize(cdiv(A.R, blockSize.x));

    spmv_ell_kernel<T><<<gridSize, blockSize>>>(A, x, y);
    
    CHECK_LAST_CUDA_ERROR();    
}

template <typename T>
T compute_torch_mv(T A, T x) {
    T ans = torch::matmul(A, x);
    return ans;
}

template <typename T>
void run_engine(float sparsity_ratio, unsigned int R, unsigned int C, float abs_tol, double rel_tol) {
    SparseMatrix<T> sparse_matrix = generate_sparse_matrix<T>(sparsity_ratio, R, C);

    ELLMatrix<T> A_h = sparse_to_ell<T>(sparse_matrix);
    T* x_h = nullptr;
    T* y_h = nullptr;

    CHECK_CUDA_ERROR(hipHostMalloc(&x_h, C*sizeof(T)));
    CHECK_CUDA_ERROR(hipHostMalloc(&y_h, R*sizeof(T)));

    random_initialize_array(x_h, C, 1337);
    std::fill(y_h, y_h+R, static_cast<T>(0));

    ELLMatrix<T> A_d;
    T *x_d, *y_d;

    move_ell_matrix_to_device(A_h, A_d);

    CHECK_CUDA_ERROR(hipMalloc(&x_d, C*sizeof(T)));
    CHECK_CUDA_ERROR(hipMalloc(&y_d, R*sizeof(T)));

    CHECK_CUDA_ERROR(hipMemcpy(x_d, x_h, C*sizeof(T), hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(y_d, y_h, R*sizeof(T), hipMemcpyHostToDevice));

    spmv_ell<T>(A_d, x_d, y_d);

    CHECK_CUDA_ERROR(hipMemcpy(y_h, y_d, R*sizeof(T), hipMemcpyDeviceToHost));
    // print_array<T>(y_h, R, "SpMV output");

    auto options = torch::TensorOptions().dtype(torch::kFloat32).requires_grad(false);
    torch::Tensor A_t = torch::from_blob(sparse_matrix.mat, {R, C}, options).clone().cuda();
    torch::Tensor x_t = torch::from_blob(x_h, {C}, options).clone().cuda();
    torch::Tensor y_cuda = torch::from_blob(y_h, {R}, options).clone();

    torch::Tensor y_t = compute_torch_mv<torch::Tensor>(A_t, x_t).cpu();

    // std::cout<<"From CUDA "<<y_cuda<<std::endl;
    // std::cout<<"From Torch "<<y_t<<std::endl;

    std::cout   << "CUDA vs Torch allclose: "
                << (torch::allclose(y_cuda, y_t, abs_tol, rel_tol) ? "true" : "false")
                << std::endl;

    CHECK_CUDA_ERROR(hipFree(A_d.rowPtrs));
    CHECK_CUDA_ERROR(hipFree(A_d.colIdx));
    CHECK_CUDA_ERROR(hipFree(A_d.value));
    CHECK_CUDA_ERROR(hipFree(x_d));
    CHECK_CUDA_ERROR(hipFree(y_d));
    free(A_h.rowPtrs);
    free(A_h.colIdx);
    free(A_h.value);
    CHECK_CUDA_ERROR(hipHostFree(x_h));
    CHECK_CUDA_ERROR(hipHostFree(y_h));
}

int main() {
    unsigned int R = ROWS;
    unsigned int C = COLUMNS;
    float sparsity_ratio = SPARSITY_RATIO;

    float abs_tol = 1.0e-3f;
    double rel_tol = 1.0e-2f;

    run_engine<float>(sparsity_ratio, R, C, abs_tol, rel_tol);

    return 0;
}