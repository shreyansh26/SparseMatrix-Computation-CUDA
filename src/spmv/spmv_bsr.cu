#include "hip/hip_runtime.h"
#include <iostream>
#include <torch/torch.h>

#include "sparse_matrix_utils.hpp"
#include "cuda_utils.hpp"

#define ROWS            16384
#define COLUMNS         16384
#define SPARSITY_RATIO  0.2
#define BLOCK_SIZE      1024

template <typename T>
void move_bsr_matrix_to_device(BSRMatrix<T>& h_matrix, BSRMatrix<T>& d_matrix) {
    unsigned int R_b = (h_matrix.R + h_matrix.block_size - 1) /  h_matrix.block_size;
    // Allocate memory on the device
    CHECK_CUDA_ERROR(hipMalloc(&d_matrix.rowPtrs, (R_b+1) * sizeof(unsigned int)));
    CHECK_CUDA_ERROR(hipMalloc(&d_matrix.colIdx, h_matrix.size_colIdx * sizeof(unsigned int)));
    CHECK_CUDA_ERROR(hipMalloc(&d_matrix.value, h_matrix.size_value * sizeof(T)));

    // Set dimensions
    d_matrix.R = h_matrix.R;
    d_matrix.C = h_matrix.C;
    d_matrix.num_nonzero = h_matrix.num_nonzero;
    d_matrix.size_colIdx = h_matrix.size_colIdx;
    d_matrix.size_value = h_matrix.size_value;
    d_matrix.block_size = h_matrix.block_size;

    // Copy data from host to device
    CHECK_CUDA_ERROR(hipMemcpy(d_matrix.rowPtrs, h_matrix.rowPtrs, (R_b+1) * sizeof(unsigned int), hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(d_matrix.colIdx, h_matrix.colIdx, h_matrix.size_colIdx * sizeof(unsigned int), hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(d_matrix.value, h_matrix.value, h_matrix.size_value * sizeof(T), hipMemcpyHostToDevice));
}

inline unsigned int cdiv(unsigned int a, unsigned int b) {
    return (a + b - 1)/b;
}

// Kernel that processes multiple rows per thread block
template <typename T>
__global__ void spmv_bsr_kernel(BSRMatrix<T> A, T* x, T* y) {
    unsigned int b = A.block_size;
    unsigned int R_b = (A.R + b - 1) / b;
    unsigned int block_row = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int thread_row = threadIdx.y;

    if(block_row < R_b) {
        for(unsigned int idx = A.rowPtrs[block_row]; idx < A.rowPtrs[block_row + 1]; idx++) {
            unsigned int block_col = A.colIdx[idx];
            T* block = &A.value[idx * b * b];

            T temp_sum = 0.0f;
            for(unsigned int j = 0; j < b; j++) {
                unsigned int col = block_col * b + j;
                if(col < A.C) {
                    temp_sum += block[thread_row * b + j] * x[col];
                }
            }

            unsigned int row = block_row * b + thread_row;
            if(row < A.R) {
                atomicAdd(&y[row], temp_sum);
            }
        }
    }
}

template <typename T>
void spmv_bsr(BSRMatrix<T> A, T* x, T* y) {
    unsigned int b = A.block_size;
    unsigned int R_b = (A.R + b - 1) / b;
    dim3 blockSize(32, b);
    dim3 gridSize(cdiv(R_b, blockSize.x));

    spmv_bsr_kernel<T><<<gridSize, blockSize>>>(A, x, y);
    
    CHECK_LAST_CUDA_ERROR();    
}

// Kernel that assigns one thread block per BSR block
template <typename T>
__global__ void spmv_bsr_block_kernel(BSRMatrix<T> A, T* x, T* y) {
    unsigned int b = A.block_size;
    unsigned int block_idx = blockIdx.x;
    unsigned int thread_row = threadIdx.x;

    unsigned int block_row = 0;
    unsigned int left = 0, right = (A.R + b - 1) / b;

    // Linear search to find the block_row
    // while(block_idx >= A.rowPtrs[block_row + 1]) {
    //     block_row++;
    // }
    
    // Binary search to find the block_row
    while(left < right) {
        unsigned int mid = (left + right) / 2;
        if(block_idx < A.rowPtrs[mid]) {
            right = mid;
        }
        else if(block_idx >= A.rowPtrs[mid + 1]) {
            left = mid + 1;
        }
        else {
            block_row = mid;
            break;
        }
    }
    if (left == right) 
        block_row = left;

    unsigned int block_col = A.colIdx[block_idx];
    T* block = &A.value[block_idx * b * b];

    if(thread_row < b) {
        T temp_sum = 0.0f;
        for(unsigned int j = 0; j < b; j++) {
            unsigned int col = block_col * b + j;
            if(col < A.C) {
                temp_sum += block[thread_row * b + j] * x[col];
            }
        }

        unsigned int row = block_row * b + thread_row;
        if(row < A.R) {
            atomicAdd(&y[row], temp_sum);
        }
    }
}

template <typename T>
void spmv_bsr_block(BSRMatrix<T> A, T* x, T* y) {
    unsigned int b = A.block_size;
    unsigned int num_blocks = A.size_value / (b * b);
    dim3 blockSize(b);
    dim3 gridSize(num_blocks);

    spmv_bsr_block_kernel<T><<<gridSize, blockSize>>>(A, x, y);
    
    CHECK_LAST_CUDA_ERROR();    
}

template <typename T>
T compute_torch_mv(T A, T x) {
    T ans = torch::matmul(A, x);
    return ans;
}

template <typename T>
void spmv_bsr_cpu(BSRMatrix<T> bsr, T* x, T* y) {
    unsigned int b = bsr.block_size;
    unsigned int R_b = (bsr.R + b - 1)/b;

    for(unsigned int block_row = 0; block_row < R_b; block_row++) {
        for(unsigned int idx = bsr.rowPtrs[block_row]; idx < bsr.rowPtrs[block_row+1]; idx++) {
            unsigned int block_col = bsr.colIdx[idx];
            T* block = &bsr.value[idx * b * b];

            for(unsigned int i=0; i<b; i++) {
                unsigned int row = block_row * b + i;
                if(row >= bsr.R)
                    continue;
        
                for(unsigned j=0; j<b; j++) {
                    unsigned int col = block_col * b + j;
                    if(col < bsr.C) {
                        y[row] += block[i * b + j] * x[col];
                    }
                }
            }
        }
    }
}

template <typename T>
void run_engine(float sparsity_ratio, unsigned int R, unsigned int C, float abs_tol, double rel_tol) {
    SparseMatrix<T> sparse_matrix = generate_sparse_matrix<T>(sparsity_ratio, R, C);
    
    unsigned int bsr_block_size = 32;
    unsigned int R_b = (sparse_matrix.R + bsr_block_size - 1) / bsr_block_size;
    unsigned int C_b = (sparse_matrix.C + bsr_block_size - 1) / bsr_block_size;

    BSRMatrix<T> A_h = sparse_to_bsr<T>(sparse_matrix, bsr_block_size);

    T* x_h = nullptr;
    T* y_h = nullptr;
    T* y_h_cpu_ref = nullptr;

    CHECK_CUDA_ERROR(hipHostMalloc(&x_h, C*sizeof(T)));
    CHECK_CUDA_ERROR(hipHostMalloc(&y_h, R*sizeof(T)));
    CHECK_CUDA_ERROR(hipHostMalloc(&y_h_cpu_ref, R*sizeof(T)));

    random_initialize_array(x_h, C, 1337);
    std::fill(y_h, y_h+R, static_cast<T>(0));
    std::fill(y_h_cpu_ref, y_h_cpu_ref+R, static_cast<T>(0));

    spmv_bsr_cpu<T>(A_h, x_h, y_h_cpu_ref);
    // print_array<T>(y_h_cpu_ref, R, "SpMV output CPU");

    BSRMatrix<T> A_d;
    T *x_d, *y_d;

    move_bsr_matrix_to_device(A_h, A_d);

    CHECK_CUDA_ERROR(hipMalloc(&x_d, C*sizeof(T)));
    CHECK_CUDA_ERROR(hipMalloc(&y_d, R*sizeof(T)));

    CHECK_CUDA_ERROR(hipMemcpy(x_d, x_h, C*sizeof(T), hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(y_d, y_h, R*sizeof(T), hipMemcpyHostToDevice));

    spmv_bsr<T>(A_d, x_d, y_d);
    // spmv_bsr_block<T>(A_d, x_d, y_d);

    CHECK_CUDA_ERROR(hipMemcpy(y_h, y_d, R*sizeof(T), hipMemcpyDeviceToHost));
    // print_array<T>(y_h, R, "SpMV output CUDA");

    auto options = torch::TensorOptions().dtype(torch::kFloat32).requires_grad(false);
    torch::Tensor A_t = torch::from_blob(sparse_matrix.mat, {R, C}, options).clone().cuda();
    torch::Tensor x_t = torch::from_blob(x_h, {C}, options).clone().cuda();
    torch::Tensor y_cpu = torch::from_blob(y_h_cpu_ref, {R}, options).clone();
    torch::Tensor y_cuda = torch::from_blob(y_h, {R}, options).clone();

    torch::Tensor y_t = compute_torch_mv<torch::Tensor>(A_t, x_t).cpu();

    // std::cout<<"From CPU "<<y_cpu<<std::endl;
    // std::cout<<"From CUDA "<<y_cuda<<std::endl;
    // std::cout<<"From Torch "<<y_t<<std::endl;

    std::cout   << "CPU vs CUDA allclose: "
                << (all_close<T>(y_h_cpu_ref, y_h, sparse_matrix.R, abs_tol, rel_tol) ? "true" : "false")
                << std::endl;

    std::cout   << "CPU vs Torch allclose: "
                << (torch::allclose(y_cpu, y_t, abs_tol, rel_tol) ? "true" : "false")
                << std::endl;

    std::cout   << "CUDA vs Torch allclose: "
                << (torch::allclose(y_cuda, y_t, abs_tol, rel_tol) ? "true" : "false")
                << std::endl;

    CHECK_CUDA_ERROR(hipFree(A_d.rowPtrs));
    CHECK_CUDA_ERROR(hipFree(A_d.colIdx));
    CHECK_CUDA_ERROR(hipFree(A_d.value));
    CHECK_CUDA_ERROR(hipFree(x_d));
    CHECK_CUDA_ERROR(hipFree(y_d));
    free(A_h.rowPtrs);
    free(A_h.colIdx);
    free(A_h.value);
    CHECK_CUDA_ERROR(hipHostFree(x_h));
    CHECK_CUDA_ERROR(hipHostFree(y_h));
}

int main() {
    unsigned int R = ROWS;
    unsigned int C = COLUMNS;
    float sparsity_ratio = SPARSITY_RATIO;

    float abs_tol = 1.0e-3f;
    double rel_tol = 1.0e-2f;

    run_engine<float>(sparsity_ratio, R, C, abs_tol, rel_tol);

    return 0;
}