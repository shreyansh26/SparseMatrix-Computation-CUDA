#include "hip/hip_runtime.h"
#include <iostream>
#include <torch/torch.h>

#include "sparse_matrix_utils.hpp"
#include "cuda_utils.hpp"

#define ROWS            4096
#define COLUMNS         2048
#define SPARSITY_RATIO  0.2
#define BLOCK_SIZE      32
#define DENSE_ROWS      1024

template <typename T>
void move_csc_matrix_to_device(CSCMatrix<T>& h_matrix, CSCMatrix<T>& d_matrix) {
    // Allocate memory on the device
    CHECK_CUDA_ERROR(hipMalloc(&d_matrix.colPtrs, (h_matrix.C+1) * sizeof(unsigned int)));
    CHECK_CUDA_ERROR(hipMalloc(&d_matrix.rowIdx, h_matrix.num_nonzero * sizeof(unsigned int)));
    CHECK_CUDA_ERROR(hipMalloc(&d_matrix.value, h_matrix.num_nonzero * sizeof(T)));

    // Set dimensions
    d_matrix.R = h_matrix.R;
    d_matrix.C = h_matrix.C;
    d_matrix.num_nonzero = h_matrix.num_nonzero;

    // Copy data from host to device
    CHECK_CUDA_ERROR(hipMemcpy(d_matrix.colPtrs, h_matrix.colPtrs, (h_matrix.C+1) * sizeof(unsigned int), hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(d_matrix.rowIdx, h_matrix.rowIdx, h_matrix.num_nonzero * sizeof(unsigned int), hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(d_matrix.value, h_matrix.value, h_matrix.num_nonzero * sizeof(T), hipMemcpyHostToDevice));
}

template <typename T>
__global__ void spmm_csc_dds_kernel(CSCMatrix<T> A, T* B, T* C, unsigned int N) {
    // A -> sparse matrix -> R x C
    // B -> dense matrix -> N x R
    // C -> dense matrix -> C = B @ A -> N x C
    unsigned int c = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int r = blockIdx.y * blockDim.y + threadIdx.y;

    if(c < A.C and r < N) {
        T sum = 0.0f;
        unsigned int col_start = A.colPtrs[c];
        unsigned int col_end = A.colPtrs[c+1];

        for(unsigned int i=col_start; i<col_end; i++) {
            unsigned int r_idx = A.rowIdx[i];
            T val = A.value[i];
            sum += B[r*A.R + r_idx] * val;
        }
        atomicAdd(&C[r*A.C + c], sum);
    }
}

inline unsigned int cdiv(unsigned int a, unsigned int b) {
    return (a + b - 1) / b;
}

template <typename T>
void spmm_csc_dds(CSCMatrix<T> A, T* B, T* C, int N) {
    // A -> sparse matrix -> R x C
    // B -> dense matrix -> N x R
    // C -> dense matrix -> C = B @ A -> N x C
    dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE);
    dim3 gridSize(cdiv(A.C, blockSize.x), cdiv(N, blockSize.y));

    spmm_csc_dds_kernel<T><<<gridSize, blockSize>>>(A, B, C, N);
    
    CHECK_LAST_CUDA_ERROR();    
}

template <typename T>
T compute_torch_mm(T A, T B) {
    T ans = torch::matmul(A, B);
    return ans;
}

template <typename T>
void run_engine(float sparsity_ratio, unsigned int R, unsigned int C, unsigned int N, float abs_tol, double rel_tol) {
    // A -> sparse matrix -> R x C
    // B -> dense matrix -> N x R
    // C -> dense matrix -> C = B @ A -> N x C
    SparseMatrix<T> sparse_matrix = generate_sparse_matrix<T>(sparsity_ratio, R, C);
    
    CSCMatrix<T> A_h = sparse_to_csc<T>(sparse_matrix);
    T* B_h = nullptr;
    T* C_h = nullptr;

    CHECK_CUDA_ERROR(hipHostMalloc(&B_h, N * R * sizeof(T)));
    CHECK_CUDA_ERROR(hipHostMalloc(&C_h, N * C * sizeof(T)));

    random_initialize_array(B_h, N * R, 1337);
    std::fill(C_h, C_h + N * C, static_cast<T>(0));

    CSCMatrix<T> A_d;
    T *B_d, *C_d;

    move_csc_matrix_to_device(A_h, A_d);

    CHECK_CUDA_ERROR(hipMalloc(&B_d, N * R * sizeof(T)));
    CHECK_CUDA_ERROR(hipMalloc(&C_d, N * C * sizeof(T)));

    CHECK_CUDA_ERROR(hipMemcpy(B_d, B_h, N * R * sizeof(T), hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(C_d, C_h, N * C * sizeof(T), hipMemcpyHostToDevice));

    std::cout<<"Starting kernel"<<std::endl;
    spmm_csc_dds<T>(A_d, B_d, C_d, N);
    std::cout<<"Kernel done"<<std::endl;

    CHECK_CUDA_ERROR(hipMemcpy(C_h, C_d, N * C * sizeof(T), hipMemcpyDeviceToHost));

    auto options = torch::TensorOptions().dtype(torch::kFloat32).requires_grad(false);
    torch::Tensor A_t = torch::from_blob(sparse_matrix.mat, {R, C}, options).clone().cuda();
    torch::Tensor B_t = torch::from_blob(B_h, {N, R}, options).clone().cuda();
    torch::Tensor C_cuda = torch::from_blob(C_h, {N, C}, options).clone();

    torch::Tensor C_t = compute_torch_mm<torch::Tensor>(B_t, A_t).cpu();

    std::cout << "CUDA vs Torch allclose: "
              << (torch::allclose(C_cuda, C_t, abs_tol, rel_tol) ? "true" : "false")
              << std::endl;

    CHECK_CUDA_ERROR(hipFree(A_d.colPtrs));
    CHECK_CUDA_ERROR(hipFree(A_d.rowIdx));
    CHECK_CUDA_ERROR(hipFree(A_d.value));
    CHECK_CUDA_ERROR(hipFree(B_d));
    CHECK_CUDA_ERROR(hipFree(C_d));
    free(A_h.colPtrs);
    free(A_h.rowIdx);
    free(A_h.value);
    CHECK_CUDA_ERROR(hipHostFree(B_h));
    CHECK_CUDA_ERROR(hipHostFree(C_h));
}

int main() {
    unsigned int R = ROWS;
    unsigned int C = COLUMNS;
    unsigned int N = DENSE_ROWS;
    float sparsity_ratio = SPARSITY_RATIO;

    float abs_tol = 1.0e-3f;
    double rel_tol = 1.0e-2f;

    run_engine<float>(sparsity_ratio, R, C, N, abs_tol, rel_tol);

    return 0;
}
